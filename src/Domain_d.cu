#include "hip/hip_runtime.h"

namespace FVM{
Domain_d::SetDimension(const int &nc, const int &cc, const int &fc){
  
  m_node_count = nc;
  m_cell_count = cc;
  m_face_count = fc;
  
  //// CELL ALLOCATION
  hipMalloc((void **)&c_PK,            m_cell_count * 6 * sizeof (double)); // PIOLA  KIRCHOFF
  hipMalloc((void **)&m_c_face_count,  m_cell_count * sizeof (unsigned int)); // PIOLA  KIRCHOFF
  hipMalloc((void **)&m_c_face,        m_cell_count * 4 * sizeof (unsigned int)); // PIOLA  KIRCHOFF
// hipMalloc((void **)&m_c_offset,      m_cell_count * sizeof (unsigned int)); // PIOLA  KIRCHOFF
  //OFSET NOT SET

  //FACE ALLOCATION
  hipMalloc((void **)&m_Af,        m_face_count * sizeof (double)); // PIOLA  KIRCHOFF
	hipMalloc((void **)&m_e_pn,      m_face_count * sizeof (double3));											  //Vector normalizado entre baricentros


}


void Domain_d::AddBoxLength(double3 const & V, double3 const & L, const double &r,const bool &red_int){

    double3 Xp;
    int p, nnodz;

    int nel[3];
    m_dim = 3;
    if (L.z > 0.0) m_dim = 3;
    
    
    nel[0] = (int)(L.x/(2.0*r));
    nel[1] = (int)(L.y/(2.0*r));
    cout << "Nel x: "<<nel[0]<<", y "<<nel[1]<<endl;
    
    m_gp_count = 1;
    if (m_dim == 2){
      nel[2] = 1;
      m_nodxelem = 4;
      if (!red_int) m_gp_count = 4;
    } else {
      nel[2] = (int)(L.z/(2.0*r));
      m_nodxelem = 8;
      if (!red_int) m_gp_count = 8; 
    }
    

    Xp.z = V.z ;
    

    // write (*,*) "Creating Mesh ...", "Elements ", neL.y, ", ",neL.z
  int nc = (nel[0] +1) * (nel[1]+1) * (nel[2]+1);
  int ne = nel[0]*nel[1]*nel[2];
  cout << "Mesh created. Element count: "<< nel[0]<<", "<<nel[1]<<", "<<nel[2]<<endl;
  
  //thisAllocateNodes((nel[0] +1) * (nel[1]+1) * (nel[2]+1));
    // print *, "Element count in XYZ: ", nel(:)
    // write (*,*) "Box Node count ", node_count
  //FACE CALCULATION
  unsigned int nf;
	fc =  nel[0]*nel[1]*(nel[2]+1)+
        nel[0]*nel[2]*(nel[1]+1)+
        nel[1]*nel[2]*(nel[0]+1);  
  this->SetDimension(nc,ne, fc);	 //AFTER CREATING DOMAIN

  cout << "Mesh generated. Node count: " << nc<<". Element count: "<<ne<<". Face count: "<<fc<<endl;
  cout << "Dimension is: "<<m_dim<<endl;
  //SPH::Domain	dom;
	//double3 *x =  (double3 *)malloc(dom.Particles.size());
	double3 *x_H =  new double3 [m_node_count];


	//int size = dom.Particles.size() * sizeof(double3);
	cout << "Copying to device..."<<endl;
    
    cout << "Box Particle Count is " << m_node_count <<endl;
    p = 0;
    for (int k = 0; k < (nel[2] +1);k++) {
      Xp.y = V.y;
      for (int j = 0; j < (nel[1] +1);j++){
        Xp.x = V.x;
        for (int i = 0; i < (nel[0] +1);i++){
					//m_node.push_back(new Node(Xp));
					x_H[p] = Xp;
          //nod%x(p,:) = Xp(:);
          cout << "node " << p <<"X: "<<Xp.x<<"Y: "<<Xp.y<<"Z: "<<Xp.z<<endl;
          p++;
          Xp.x = Xp.x + 2.0 * r;
        }
        Xp.y = Xp.y + 2.0 * r;
      }// 
      Xp.z = Xp.z + 2 * r;

    //cout <<"m_node size"<<m_node.size()<<endl;
    } 
		hipMemcpy(this->x, x_H, sizeof(double3) * m_node_count, hipMemcpyHostToDevice);    

    // !! ALLOCATE ELEMENTS
    // !! DIMENSION = 2
    int gp = 1;
    if (m_dim == 2) {
      // if (redint .eqv. .False.) then
        // gp = 4
      // end if 
      //call AllocateElements(neL.y * neL.z,gp) !!!!REDUCED INTEGRATION
    } else {
      // if (redint .eqv. .False.) then
        // gp = 8
      // end if 
      // call AllocateElements(neL.y * neL.z*nel(3),gp) 
    }

		unsigned int *elnod_h = new unsigned int [m_elem_count * m_nodxelem]; //Flattened
    
		int ex, ey, ez;
		std::vector <int> n;
    if (m_dim == 2) {
			n.resize(4);
      int ei = 0;
      for (int ey = 0; ey < nel[1];ey++){
        for (int ex = 0; ex < nel[0];ex++){
        int iv[4];
        elnod_h[ei  ] = (nel[0]+1)*ey + ex;        elnod_h[ei+1] = (nel[0]+1)*ey + ex+1;
        elnod_h[ei+2] = (nel[0]+1)*(ey+1) + ex+1;  elnod_h[ei+3] = (nel[0]+1)*(ey+1) + ex;
			
				 for (int i=0;i<m_nodxelem;i++)cout << elnod_h[ei+i]<<", ";
					cout << "Nel x : "<<nel[0]<<endl;
					cout << "nodes "<<endl;
					ei += m_nodxelem;
					 }
      } 
    } else { //dim: 3
      int ei = 0;
      int nnodz = (nel[0]+1)*(nel[1]+1);
      for (int ez = 0; ez < nel[2];ez++)
      for (int ey = 0; ey < nel[1];ey++){
        for (int ex = 0; ex < nel[0];ex++){
          
          int iv[8];
          int nb1 = nnodz*ez + (nel[0]+1)*ey + ex;
          int nb2 = nnodz*ez + (nel[0]+1)*(ey+1) + ex;
          elnod_h[ei  ] = nb1;
          elnod_h[ei+1] = nb1+1;
          elnod_h[ei+2] = nb2+1;
          elnod_h[ei+3] = nb2;
          elnod_h[ei+4] = nb1 + nnodz*(ez+1);
          elnod_h[ei+5] = nb1 + nnodz*(ez+1) + 1;
          elnod_h[ei+6] = nb2 + nnodz*(ez+1) + 1;
          elnod_h[ei+7] = nb2 + nnodz*(ez+1);
          
          for (int i=0;i<8;i++)
            cout << elnod_h[ei + i]<<", ";
          cout <<endl;

          // elem%elnod(i,:) = [ nnodz*ez + (nel(1)+1)*ey + ex+1,nnodz*ez + (nel(1)+1)*ey + ex+2, &
                              // nnodz*ez + (nel(1)+1)*(ey+1)+ex+2,nnodz*ez + (nel(1)+1)*(ey+1)+ex+1, &
                              // nnodz*(ez + 1) + (nel(1)+1)*ey + ex+1,nnodz*(ez + 1) + (nel(1)+1)*ey + ex+2, &
                              // nnodz*(ez + 1) + (nel(1)+1)*(ey+1)+ex+2,nnodz*(ez + 1)+ (nel(1)+1)*(ey+1)+ex+1];
        // cout << i[]
						// n[0]= m_node[iv[0]];
						// n[1]= m_node[(nel[0]+1)*ey + ex+1];
						// n[2]= m_node[(nel[0]+1)*(ey+1)+ex+1];
						// n[3]= m_node[(nel[0]+1)*(ey+1)+ex];
            cout << "Nel x : "<<nel[0]<<endl;
           cout << "nodes "<<endl;
           
           for (int i=0;i<m_nodxelem;i++)cout << elnod_h[ei+i]<<", ";
           ei += m_nodxelem;
						 //m_element.push_back(new El4N2DPE(n));
																							// m_node[(nel[0]+1)*ey + ex+1],
																							// m_node[(nel[0]+1)*(ey+1)+ex+1],
																							// m_node[(nel[0]+1)*(ey+1)+ex]
																							// );
              //elem%elnod(i,:)=[(neL.y+1)*ey + ex+1,(neL.y+1)*ey + ex+2,(neL.y+1)*(ey+1)+ex+2,(neL.y+1)*(ey+1)+ex+1]         
              //print *, "Element ", i , "Elnod", elem%elnod(i,:) 
					 }
      } 

		}//if dim 

    hipMalloc((void **)&m_elnod, m_elem_count * m_nodxelem * sizeof (int));		
		hipMemcpy(this->m_elnod, elnod_h, sizeof(unsigned int) * m_elem_count * m_nodxelem, hipMemcpyHostToDevice);    
    
    hipMalloc(&m_jacob,m_elem_count * sizeof(Matrix ));
    
	
		delete [] elnod_h;
}

};